#include "hip/hip_runtime.h"
#include "kernel.h"
#include <iostream>
#include <opencv2/core.hpp>
#include <opencv2/imgcodecs.hpp>
#include <stdio.h>
#include <complex>
#include "math.h"
#include <hip/hip_complex.h>
//compute numer of iterations to diverge
__device__ int mandelbrotIterations(const hipDoubleComplex &z0, const int max){
    hipDoubleComplex z = z0;
    for (int t = 0; t < max; t++){

        if( (hipCreal(z)*hipCreal(z) + hipCimag(z)*hipCimag(z) ) > 4.0f){

           return t;

        }
        z = hipCadd(hipCmul(z,z), z0);

    }
    return max;
}
__device__ int mandelbrotSet(const hipDoubleComplex &z0, const int maxIter=500){
   //does it diverge?
    int iterations = mandelbrotIterations(z0, maxIter);
    //avoid division by zero
    if(maxIter - iterations == 0){
        return 0;
    }
    //rescale value to 8 bits (CV_U8)
    return lrint(sqrt(iterations / (float) maxIter) * 255);
}
__global__ void kernel(unsigned char *d_output, int rows, int cols,float x1, float y1, float scaleX, float scaleY){

  // get correspondig coordinates from grid indexes
  int c = blockIdx.x*blockDim.x + threadIdx.x;
  int r = blockIdx.y*blockDim.y + threadIdx.y;
  const int i = r*cols + c;

  // check image bounds
  if( (r>=rows) || (c>=cols) ){
    return;
  }

  //perform operation
  float x0= c/scaleX + x1;
  float y0= r/scaleY +y1;
  hipDoubleComplex z0 = make_hipDoubleComplex(x0, y0);
  uchar value = (uchar) mandelbrotSet(z0);
  d_output[i]= value;
}

void wrapper_gpu(Mat output){
  unsigned char *outputPtr = (unsigned char*) output.data;
  unsigned int cols = output.cols;
  unsigned int rows = output.rows;
  float x1 = -2.1f;
  float x2 =  0.6f;
  float y1 = -1.2f;
  float y2 =  1.2f;
  float scaleX = output.cols / (x2 - x1);
  float scaleY = output.rows / (y2 - y1);

  //block dimensions (threads)
  int Tx = 32;
  int Ty = 32;

  //grid size dimensions (blocks)
  int Bx = (Tx + rows -1)/Tx;
  int By = (Ty + cols -1)/Ty;

  // declare pointers to device memory
  unsigned char *d_in  = 0;
  unsigned char *d_out = 0;

  // allocate memory in device
  hipMalloc(&d_in, cols*rows*sizeof(unsigned char));
  hipMalloc(&d_out, cols*rows*sizeof(unsigned char));

  //prepare kernel lauch dimensions
  const dim3 blockSize = dim3(Tx, Ty);
  const dim3 gridSize= dim3(Bx, By);

  // launch kernel in GPU
  kernel<<<gridSize, blockSize>>>(d_out, rows, cols, x1,y1, scaleX, scaleY);

  // copy output from device to host
  hipMemcpy(outputPtr, d_out, rows*cols*sizeof(unsigned char), hipMemcpyDeviceToHost);

  // free the memory allocated for device arrays
  hipFree(d_in);
  hipFree(d_out);

}